
#include <hip/hip_runtime.h>
// #include "lbm.cuh"

// void initialize_cuda(float* distances, float* directional_densities, float* directional_densities_tmp, float* densities, float* ux, float* uy, int* walls) {
//     float* cuda_distances, cuda_directional_densities, cuda_directional_densities_tmp, cuda_densities, cuda_ux, cuda_uy;
//     int* cuda_walls;
//     cudaMalloc(&cuda_distances, totalPoints*sizeof(float));
//     cudaMalloc(&cuda_directional_densities, totalPoints*9*sizeof(float));
//     cudaMalloc(&cuda_directional_densities_tmp, totalPoints*9*sizeof(float));
//     cudaMalloc(&cuda_densities, totalPoints*sizeof(float));
//     cudaMalloc(&cuda_ux, totalPoints*sizeof(float));
//     cudaMalloc(&cuda_uy, totalPoints*sizeof(float));
//     cudaMalloc(&cuda_walls, totalPoints*sizeof(int));
    
//     cudaMemcpy(cuda_directional_densities, directional_densities, totalPoints*9*sizeof(float), cudaMemcpyHostToDevice)
// }